#include "hip/hip_runtime.h"
/**
 * BMW512
 */
extern "C" {
#include "sph/sph_bmw.h"
}
#include "miner.h"
#include "cuda_helper.h"
#include <unistd.h>

#define NBN 2

static uint32_t *d_resNonce[MAX_GPUS];
static uint32_t *h_resNonce[MAX_GPUS];
extern void quark_bmw512_cpu_init(int thr_id, uint32_t threads);
extern void quark_bmw512_cpu_setBlock_80(void *pdata);
void quark_bmw512_cpu_hash_80_final(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_resNonce, const uint64_t target);


extern "C" void bmw512_hash(void *state, const void *input) {
	sph_bmw512_context ctx_bmw;
	unsigned char hash[64];

	sph_bmw512_init(&ctx_bmw);
	sph_bmw512(&ctx_bmw, input, 80);
	sph_bmw512_close(&ctx_bmw, hash);
	memcpy(state, hash, 32);
}


static bool init[MAX_GPUS] = { 0 };


extern "C" int scanhash_bmw512(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done) {
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	uint32_t endiandata[20];

	if (opt_benchmark) ptarget[7] = 0x00ff;

	for (int k=0; k < 20; k++) be32enc(&endiandata[k], pdata[k]);

	uint32_t throughput =  cuda_default_throughput(thr_id, 1 << 28);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (!init[thr_id]) {
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}

		CUDA_SAFE_CALL(hipMalloc(&d_resNonce[thr_id], NBN * sizeof(uint32_t)));
		h_resNonce[thr_id] = (uint32_t*) malloc(NBN * sizeof(uint32_t));
		if(h_resNonce[thr_id] == NULL){
			gpulog(LOG_ERR,thr_id,"Host memory allocation failed");
			exit(EXIT_FAILURE);
		}
		quark_bmw512_cpu_init(thr_id, throughput);
		cuda_check_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	quark_bmw512_cpu_setBlock_80((void*)endiandata);
	hipMemset(d_resNonce[thr_id], 0xff, NBN*sizeof(uint32_t));
	*hashes_done = 0;

	do {
		quark_bmw512_cpu_hash_80_final(thr_id, throughput, pdata[19], d_resNonce[thr_id], *(uint64_t*)&ptarget[6]);
		hipMemcpy(h_resNonce[thr_id], d_resNonce[thr_id], NBN*sizeof(uint32_t), hipMemcpyDeviceToHost);
		*hashes_done += throughput;

		if (h_resNonce[thr_id][0] != UINT32_MAX) {
			const uint32_t Htarg = ptarget[7];
			const uint32_t startNounce = pdata[19];
			uint32_t _ALIGN(64) vhash[8];

			be32enc(&endiandata[19], startNounce + h_resNonce[thr_id][0]);
			bmw512_hash(vhash, endiandata);
			if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work->nonces[0] = startNounce + h_resNonce[thr_id][0];
				work_set_target_ratio(work, vhash);
				if (h_resNonce[thr_id][1] != UINT32_MAX) {
					uint32_t secNonce = work->nonces[1] = startNounce + h_resNonce[thr_id][1];
					be32enc(&endiandata[19], secNonce);
					bmw512_hash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
					gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				hipMemset(d_resNonce[thr_id], 0xff, 2*sizeof(uint32_t));
				pdata[19] = startNounce + h_resNonce[thr_id][0] + 1;
				continue;
			}
		}

		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);
	return 0;
}


extern "C" void free_bmw512(int thr_id) {
	if (!init[thr_id]) return;

	hipSetDevice(device_map[thr_id]);

	free(h_resNonce[thr_id]);
	hipFree(d_resNonce[thr_id]);
	init[thr_id] = false;

	hipDeviceSynchronize();
}