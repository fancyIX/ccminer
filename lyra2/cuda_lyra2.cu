#include "hip/hip_runtime.h"
/**
 * fancyIX
 * Lyra2 (v1) cuda implementation based on djm34 work
 * tpruvot@github 2015, Nanashi 08/2016 (from 1.8-r2)
 */

 #include <stdio.h>
 #include <memory.h>
 
 #define TPB52 32
 
 #include "cuda_lyra2_sm2.cuh"
 #include "cuda_lyra2_sm5.cuh"
 
 #ifdef __INTELLISENSE__
 /* just for vstudio code colors */
 #define __CUDA_ARCH__ 520
 #endif
 
 #if !defined(__CUDA_ARCH__) ||  __CUDA_ARCH__ > 500
 
 #include "cuda_lyra2_vectors.h"
 
 #ifdef __INTELLISENSE__
 /* just for vstudio code colors */
 __device__ uint32_t __shfl(uint32_t a, uint32_t b, uint32_t c);
 #endif
 
 #define Nrow 8
 #define Ncol 8
 #define memshift 3
 
 #define BUF_COUNT 0
 
 __device__ uint2 *DMatrix;
 

 __device__ __forceinline__ void LD4SS(uint2 res[3], const int row, const int col, const int thread, const int threads)
 {
     extern __shared__ uint2 shared_mem[];
     const int s0 = (Ncol * (row - BUF_COUNT) + col) * memshift;

	res[0] = shared_mem[((s0 + 0) * 8 + threadIdx.y) * 4 + threadIdx.x];
	res[1] = shared_mem[((s0 + 1) * 8 + threadIdx.y) * 4 + threadIdx.x];
	res[2] = shared_mem[((s0 + 2) * 8 + threadIdx.y) * 4 + threadIdx.x];
 }
 
 __device__ __forceinline__ void ST4SS(const int row, const int col, const uint2 data[3], const int thread, const int threads)
 {
     extern __shared__ uint2 shared_mem[];
     const int s0 = (Ncol * (row - BUF_COUNT) + col) * memshift;
 
     shared_mem[((s0 + 0) * 8 + threadIdx.y) * 4 + threadIdx.x] = data[0];
	 shared_mem[((s0 + 1) * 8 + threadIdx.y) * 4 + threadIdx.x] = data[1];
	 shared_mem[((s0 + 2) * 8 + threadIdx.y) * 4 + threadIdx.x] = data[2];
 }

 
 #if __CUDA_ARCH__ >= 300
 __device__ __forceinline__ uint32_t WarpShuffle(uint32_t a, uint32_t b, uint32_t c)
 {
     return __shfl(a, b, c);
 }
 
 __device__ __forceinline__ uint2 WarpShuffle(uint2 a, uint32_t b, uint32_t c)
 {
     return make_uint2(__shfl(a.x, b, c), __shfl(a.y, b, c));
 }
 
 __device__ __forceinline__ void WarpShuffle3(uint2 &a1, uint2 &a2, uint2 &a3, uint32_t b1, uint32_t b2, uint32_t b3, uint32_t c)
 {
     a1 = WarpShuffle(a1, b1, c);
     a2 = WarpShuffle(a2, b2, c);
     a3 = WarpShuffle(a3, b3, c);
 }
 
 #else
 __device__ __forceinline__ uint32_t WarpShuffle(uint32_t a, uint32_t b, uint32_t c)
 {
     extern __shared__ uint2 shared_mem[];
 
     const uint32_t thread = blockDim.x * threadIdx.y + threadIdx.x;
     uint32_t *_ptr = (uint32_t*)shared_mem;
 
     __threadfence_block();
     uint32_t buf = _ptr[thread];
 
     _ptr[thread] = a;
     __threadfence_block();
     uint32_t result = _ptr[(thread&~(c - 1)) + (b&(c - 1))];
 
     __threadfence_block();
     _ptr[thread] = buf;
 
     __threadfence_block();
     return result;
 }
 
 __device__ __forceinline__ uint2 WarpShuffle(uint2 a, uint32_t b, uint32_t c)
 {
     extern __shared__ uint2 shared_mem[];
 
     const uint32_t thread = blockDim.x * threadIdx.y + threadIdx.x;
 
     __threadfence_block();
     uint2 buf = shared_mem[thread];
 
     shared_mem[thread] = a;
     __threadfence_block();
     uint2 result = shared_mem[(thread&~(c - 1)) + (b&(c - 1))];
 
     __threadfence_block();
     shared_mem[thread] = buf;
 
     __threadfence_block();
     return result;
 }
 
 __device__ __forceinline__ void WarpShuffle3(uint2 &a1, uint2 &a2, uint2 &a3, uint32_t b1, uint32_t b2, uint32_t b3, uint32_t c)
 {
     extern __shared__ uint2 shared_mem[];
 
     const uint32_t thread = blockDim.x * threadIdx.y + threadIdx.x;
 
     __threadfence_block();
     uint2 buf = shared_mem[thread];
 
     shared_mem[thread] = a1;
     __threadfence_block();
     a1 = shared_mem[(thread&~(c - 1)) + (b1&(c - 1))];
     __threadfence_block();
     shared_mem[thread] = a2;
     __threadfence_block();
     a2 = shared_mem[(thread&~(c - 1)) + (b2&(c - 1))];
     __threadfence_block();
     shared_mem[thread] = a3;
     __threadfence_block();
     a3 = shared_mem[(thread&~(c - 1)) + (b3&(c - 1))];
 
     __threadfence_block();
     shared_mem[thread] = buf;
     __threadfence_block();
 }
 
 #endif
 
 #if __CUDA_ARCH__ > 500 || !defined(__CUDA_ARCH)
 static __device__ __forceinline__
 void Gfunc(uint2 &a, uint2 &b, uint2 &c, uint2 &d)
 {
     a += b; uint2 tmp = d; d.y = a.x ^ tmp.x; d.x = a.y ^ tmp.y;
     c += d; b ^= c; b = ROR24(b);
     a += b; d ^= a; d = ROR16(d);
     c += d; b ^= c; b = ROR2(b, 63);
 }
 #endif
 
 __device__ __forceinline__ void round_lyra(uint2 s[4])
 {
     Gfunc(s[0], s[1], s[2], s[3]);
     WarpShuffle3(s[1], s[2], s[3], threadIdx.x + 1, threadIdx.x + 2, threadIdx.x + 3, 4);
     Gfunc(s[0], s[1], s[2], s[3]);
     WarpShuffle3(s[1], s[2], s[3], threadIdx.x + 3, threadIdx.x + 2, threadIdx.x + 1, 4);
 }
 
 static __device__ __forceinline__
 void round_lyra(uint2x4* s)
 {
     Gfunc(s[0].x, s[1].x, s[2].x, s[3].x);
     Gfunc(s[0].y, s[1].y, s[2].y, s[3].y);
     Gfunc(s[0].z, s[1].z, s[2].z, s[3].z);
     Gfunc(s[0].w, s[1].w, s[2].w, s[3].w);
     Gfunc(s[0].x, s[1].y, s[2].z, s[3].w);
     Gfunc(s[0].y, s[1].z, s[2].w, s[3].x);
     Gfunc(s[0].z, s[1].w, s[2].x, s[3].y);
     Gfunc(s[0].w, s[1].x, s[2].y, s[3].z);
 }
 
 static __device__ __forceinline__
 void reduceDuplex(uint2 state[4], uint32_t thread, const uint32_t threads)
 {
     uint2 state1[3];
	 uint2 state2[3];
 

     for (int i = 0; i < Nrow; i++)
     {
         ST4SS(0, Ncol - i - 1, state, thread, threads);
 
         round_lyra(state);
     }
 
     for (int i = 0; i < Nrow; i+=2)
     {
         LD4SS(state1, 0, i, thread, threads);
		 LD4SS(state2, 0, i + 1, thread, threads);
		 #pragma unroll
         for (int j = 0; j < 3; j++)
             state[j] ^= state1[j];
 
         round_lyra(state);
 
		 #pragma unroll
         for (int j = 0; j < 3; j++)
             state1[j] ^= state[j];
		
			 #pragma unroll
         for (int j = 0; j < 3; j++)
             state[j] ^= state2[j];
 
         round_lyra(state);
 
		 #pragma unroll
         for (int j = 0; j < 3; j++)
             state2[j] ^= state[j];
         ST4SS(1, Ncol - i - 1, state1, thread, threads);
		 ST4SS(1, Ncol - (i + 1) - 1, state2, thread, threads);
     }
 }
 
 static __device__ __forceinline__
 void reduceDuplexRowSetup(const int rowIn, const int rowInOut, const int rowOut, uint2 state[4], uint32_t thread, const uint32_t threads)
 {
     uint2 state1[3], state2[3], state3[3], state4[3];

     for (int i = 0; i < Nrow; i+=2)
     {
         LD4SS(state1, rowIn, i, thread, threads);
		 LD4SS(state2, rowInOut, i, thread, threads);
		 LD4SS(state3, rowIn, i + 1, thread, threads);
		 LD4SS(state4, rowInOut, i + 1, thread, threads);
		 #pragma unroll
         for (int j = 0; j < 3; j++)
             state[j] ^= state1[j] + state2[j];
 
         round_lyra(state);
 
         #pragma unroll
         for (int j = 0; j < 3; j++)
             state1[j] ^= state[j];
 
         ST4SS(rowOut, Ncol - i - 1, state1, thread, threads);
 
         // simultaneously receive data from preceding thread and send data to following thread
         uint2 Data0 = state[0];
         uint2 Data1 = state[1];
         uint2 Data2 = state[2];
         WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);
 
         if (threadIdx.x == 0)
         {
             state2[0] ^= Data2;
             state2[1] ^= Data0;
             state2[2] ^= Data1;
         } else {
             state2[0] ^= Data0;
             state2[1] ^= Data1;
             state2[2] ^= Data2;
         }
 
         ST4SS(rowInOut, i, state2, thread, threads);

		//=====================================
		 #pragma unroll
         for (int j = 0; j < 3; j++)
             state[j] ^= state3[j] + state4[j];
 
         round_lyra(state);
 
         #pragma unroll
         for (int j = 0; j < 3; j++)
             state3[j] ^= state[j];
 
         ST4SS(rowOut, Ncol - (i + 1) - 1, state3, thread, threads);
 
         // simultaneously receive data from preceding thread and send data to following thread
         uint2 Data01 = state[0];
         uint2 Data11 = state[1];
         uint2 Data21 = state[2];
         WarpShuffle3(Data01, Data11, Data21, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);
 
         if (threadIdx.x == 0)
         {
             state4[0] ^= Data21;
             state4[1] ^= Data01;
             state4[2] ^= Data11;
         } else {
             state4[0] ^= Data01;
             state4[1] ^= Data11;
             state4[2] ^= Data21;
         }
 
         ST4SS(rowInOut, (i + 1), state4, thread, threads);
     }
 }
 
 static __device__ __forceinline__
 void reduceDuplexRowt(const int rowIn, const int rowInOut, const int rowOut, uint2 state[4], const uint32_t thread, const uint32_t threads)
 {
     for (int i = 0; i < Nrow; i+=2)
     {
         uint2 state1[3], state2[3], state3[3], state4[3];
 
         LD4SS(state1, rowIn, i, thread, threads);
         LD4SS(state2, rowInOut, i, thread, threads);
		 LD4SS(state3, rowIn, i + 1, thread, threads);
         LD4SS(state4, rowInOut, i + 1, thread, threads);
 
 #pragma unroll
         for (int j = 0; j < 3; j++)
             state[j] ^= state1[j] + state2[j];
 
         LD4SS(state1, rowOut, i, thread, threads);

         round_lyra(state);
 
         // simultaneously receive data from preceding thread and send data to following thread
         uint2 Data0 = state[0];
         uint2 Data1 = state[1];
         uint2 Data2 = state[2];
         WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);
 
         if (threadIdx.x == 0)
         {
             state2[0] ^= Data2;
             state2[1] ^= Data0;
             state2[2] ^= Data1;
         }
         else
         {
             state2[0] ^= Data0;
             state2[1] ^= Data1;
             state2[2] ^= Data2;
         }

        if (rowInOut != rowOut) {
             ST4SS(rowInOut, i, state2, thread, threads);
                 #pragma unroll
            for (int j = 0; j < 3; j++)
                state2[j] = state1[j];
         }

#pragma unroll
        for (int j = 0; j < 3; j++)
            state2[j] ^= state[j];

        ST4SS(rowOut, i, state2, thread, threads);

		 //======================================
 
 
 #pragma unroll
         for (int j = 0; j < 3; j++)
             state[j] ^= state3[j] + state4[j];
 
        LD4SS(state3, rowOut, i + 1, thread, threads);

         round_lyra(state);
 
         // simultaneously receive data from preceding thread and send data to following thread
         uint2 Data01 = state[0];
         uint2 Data11 = state[1];
         uint2 Data21 = state[2];
         WarpShuffle3(Data01, Data11, Data21, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);
 
         if (threadIdx.x == 0)
         {
             state4[0] ^= Data21;
             state4[1] ^= Data01;
             state4[2] ^= Data11;
         }
         else
         {
             state4[0] ^= Data01;
             state4[1] ^= Data11;
             state4[2] ^= Data21;
         }

         if (rowInOut != rowOut) {
             ST4SS(rowInOut, i + 1, state4, thread, threads);
                 #pragma unroll
            for (int j = 0; j < 3; j++)
                state4[j] = state3[j];
         }

#pragma unroll
        for (int j = 0; j < 3; j++)
            state4[j] ^= state[j];

        ST4SS(rowOut, i + 1, state4, thread, threads);
     }
 }
 
 static __device__ __forceinline__
 void reduceDuplexRowt_8(const int rowInOut, uint2* state, const uint32_t thread, const uint32_t threads)
 {
     uint2 state1[3], state2[3], state3[3], state4[3], last[3];
 
     LD4SS(state1, 2, 0, thread, threads);
     LD4SS(last, rowInOut, 0, thread, threads);
 
     #pragma unroll
     for (int j = 0; j < 3; j++)
         state[j] ^= state1[j] + last[j];
 
     round_lyra(state);
 
     // simultaneously receive data from preceding thread and send data to following thread
     uint2 Data0 = state[0];
     uint2 Data1 = state[1];
     uint2 Data2 = state[2];
     WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);
 
     if (threadIdx.x == 0)
     {
         last[0] ^= Data2;
         last[1] ^= Data0;
         last[2] ^= Data1;
     } else {
         last[0] ^= Data0;
         last[1] ^= Data1;
         last[2] ^= Data2;
     }
 
     if (rowInOut == 5)
     {
         #pragma unroll
         for (int j = 0; j < 3; j++)
             last[j] ^= state[j];
     }
 
	 LD4SS(state1, 2, 1, thread, threads);
	 LD4SS(state2, rowInOut, 1, thread, threads);

	 #pragma unroll
	 for (int j = 0; j < 3; j++)
		 state[j] ^= state1[j] + state2[j];

	 round_lyra(state);

     for (int i = 2; i < Nrow; i+=2)
     {
         LD4SS(state1, 2, i, thread, threads);
         LD4SS(state2, rowInOut, i, thread, threads);
		 LD4SS(state3, 2, i + 1, thread, threads);
         LD4SS(state4, rowInOut, i + 1, thread, threads);
 
         #pragma unroll
         for (int j = 0; j < 3; j++)
             state[j] ^= state1[j] + state2[j];
 
         round_lyra(state);

		 //============================
 
         #pragma unroll
         for (int j = 0; j < 3; j++)
             state[j] ^= state3[j] + state4[j];
 
         round_lyra(state);
     }
 
     #pragma unroll
     for (int j = 0; j < 3; j++)
         state[j] ^= last[j];
 }

 __constant__ uint2x4 blake2b_IV[2] = {
     0xf3bcc908lu, 0x6a09e667lu,
     0x84caa73blu, 0xbb67ae85lu,
     0xfe94f82blu, 0x3c6ef372lu,
     0x5f1d36f1lu, 0xa54ff53alu,
     0xade682d1lu, 0x510e527flu,
     0x2b3e6c1flu, 0x9b05688clu,
     0xfb41bd6blu, 0x1f83d9ablu,
     0x137e2179lu, 0x5be0cd19lu
 };
 
 __global__ __launch_bounds__(64, 1)
 void lyra2_gpu_hash_32_1(uint32_t threads, uint32_t startNounce, uint2 *g_hash)
 {
     const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
 
     if (thread < threads)
     {
         uint2x4 state[4];
 
         state[0].x = state[1].x = __ldg(&g_hash[thread + threads * 0]);
         state[0].y = state[1].y = __ldg(&g_hash[thread + threads * 1]);
         state[0].z = state[1].z = __ldg(&g_hash[thread + threads * 2]);
         state[0].w = state[1].w = __ldg(&g_hash[thread + threads * 3]);
         state[2] = blake2b_IV[0];
         state[3] = blake2b_IV[1];
 
         for (int i = 0; i<24; i++)
             round_lyra(state); //because 12 is not enough
 
         ((uint2x4*)DMatrix)[threads * 0 + thread] = state[0];
         ((uint2x4*)DMatrix)[threads * 1 + thread] = state[1];
         ((uint2x4*)DMatrix)[threads * 2 + thread] = state[2];
         ((uint2x4*)DMatrix)[threads * 3 + thread] = state[3];
     }
 }
 
 __global__
 __launch_bounds__(64, 1)
 void lyra2_gpu_hash_32_2(uint32_t threads, uint32_t startNounce, uint64_t *g_hash)
 {
     const uint32_t thread = blockDim.y * blockIdx.x + threadIdx.y;
 
     if (thread < threads)
     {
         uint2 state[4];
         state[0] = __ldg(&DMatrix[(0 * threads + thread) * blockDim.x + threadIdx.x]);
         state[1] = __ldg(&DMatrix[(1 * threads + thread) * blockDim.x + threadIdx.x]);
         state[2] = __ldg(&DMatrix[(2 * threads + thread) * blockDim.x + threadIdx.x]);
         state[3] = __ldg(&DMatrix[(3 * threads + thread) * blockDim.x + threadIdx.x]);
 
         reduceDuplex(state, thread, threads);
         reduceDuplexRowSetup(1, 0, 2, state, thread, threads);
         reduceDuplexRowSetup(2, 1, 3, state, thread, threads);
         reduceDuplexRowSetup(3, 0, 4, state, thread, threads);
         reduceDuplexRowSetup(4, 3, 5, state, thread, threads);
         reduceDuplexRowSetup(5, 2, 6, state, thread, threads);
         reduceDuplexRowSetup(6, 1, 7, state, thread, threads);

         uint32_t rowa;
         uint32_t row = 0;
         uint32_t pre = 7;
         for (int i = 0; i < 7; i++) {
            rowa = WarpShuffle(state[0].x, 0, 4) & 7;
            reduceDuplexRowt(pre, rowa, row, state, thread, threads);
            pre = row;
            row = (row + 3) % 8;
         }
         rowa = WarpShuffle(state[0].x, 0, 4) & 7;
         reduceDuplexRowt_8(rowa, state, thread, threads);
 
         DMatrix[(0 * threads + thread) * blockDim.x + threadIdx.x] = state[0];
         DMatrix[(1 * threads + thread) * blockDim.x + threadIdx.x] = state[1];
         DMatrix[(2 * threads + thread) * blockDim.x + threadIdx.x] = state[2];
         DMatrix[(3 * threads + thread) * blockDim.x + threadIdx.x] = state[3];
     }
 }
 

 __global__ __launch_bounds__(64, 1)
 void lyra2_gpu_hash_32_3(uint32_t threads, uint32_t startNounce, uint2 *g_hash)
 {
     const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;
 
     uint28 state[4];
 
     if (thread < threads)
     {
         state[0] = __ldg4(&((uint2x4*)DMatrix)[threads * 0 + thread]);
         state[1] = __ldg4(&((uint2x4*)DMatrix)[threads * 1 + thread]);
         state[2] = __ldg4(&((uint2x4*)DMatrix)[threads * 2 + thread]);
         state[3] = __ldg4(&((uint2x4*)DMatrix)[threads * 3 + thread]);
 
         for (int i = 0; i < 12; i++)
             round_lyra(state);
 
         g_hash[thread + threads * 0] = state[0].x;
         g_hash[thread + threads * 1] = state[0].y;
         g_hash[thread + threads * 2] = state[0].z;
         g_hash[thread + threads * 3] = state[0].w;
 
     } //thread
 }
 #else
 #if __CUDA_ARCH__ < 500
 
 /* for unsupported SM arch */
 __device__ void* DMatrix;
 #endif
 __global__ void lyra2_gpu_hash_32_1(uint32_t threads, uint32_t startNounce, uint2 *g_hash) {}
 __global__ void lyra2_gpu_hash_32_2(uint32_t threads, uint32_t startNounce, uint64_t *g_hash) {}
 __global__ void lyra2_gpu_hash_32_3(uint32_t threads, uint32_t startNounce, uint2 *g_hash) {}
 #endif
__host__
void lyra2_cpu_init(int thr_id, uint32_t threads, uint64_t *d_matrix)
{
	// just assign the device pointer allocated in main loop
	hipMemcpyToSymbol(HIP_SYMBOL(DMatrix), &d_matrix, sizeof(uint64_t*), 0, hipMemcpyHostToDevice);
}
__host__
void lyra2_cpu_init_high_end(int thr_id, uint32_t threads, uint64_t *g_pad)
{
}

__host__
void lyra2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_hash, bool gtx750ti, bool high_end)
{
}

__host__
void lyra2_cpu_hash_32_fancyIX(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_hash, uint64_t *g_pad, bool gtx750ti, bool high_end)
{
	int dev_id = device_map[thr_id % MAX_GPUS];

	uint32_t tpb = TPB52;

	if (cuda_arch[dev_id] >= 520) tpb = TPB52;
	else if (cuda_arch[dev_id] >= 500) tpb = TPB50;
	else if (cuda_arch[dev_id] >= 200) tpb = TPB20;

	dim3 grid1((threads * 4 + 32 - 1) / 32);
	dim3 block1(4, 32 >> 2);

	dim3 grid2((threads + 64 - 1) / 64);
	dim3 block2(64);

	dim3 grid3((threads + tpb - 1) / tpb);
	dim3 block3(tpb);

	if (cuda_arch[dev_id] >= 520)
	{
		lyra2_gpu_hash_32_1 <<< grid2, block2 >>> (threads, startNounce, (uint2*)d_hash);

		    lyra2_gpu_hash_32_2 <<< grid1, block1, 24 * (8 - 0) * sizeof(uint2) * 32 >>> (threads, startNounce, d_hash);

		lyra2_gpu_hash_32_3 <<< grid2, block2 >>> (threads, startNounce, (uint2*)d_hash);
	}
	else if (cuda_arch[dev_id] >= 500)
	{
		size_t shared_mem = 0;

		if (gtx750ti)
			// suitable amount to adjust for 8warp
			shared_mem = 8192;
		else
			// suitable amount to adjust for 10warp
			shared_mem = 6144;

		lyra2_gpu_hash_32_1_sm5 <<< grid2, block2 >>> (threads, startNounce, (uint2*)d_hash);

		lyra2_gpu_hash_32_2_sm5 <<< grid1, block1, shared_mem >>> (threads, startNounce, (uint2*)d_hash);

		lyra2_gpu_hash_32_3_sm5 <<< grid2, block2 >>> (threads, startNounce, (uint2*)d_hash);
	}
	else
		lyra2_gpu_hash_32_sm2 <<< grid3, block3 >>> (threads, startNounce, d_hash);
}
